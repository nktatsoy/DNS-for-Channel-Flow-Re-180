#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <fstream>
#include <iostream>
#include <random>
#define pi 3.1415927f

using namespace std;

__global__ void bcKernel(float* Un, float* Vn, float* Wn, int Nx, int Ny, int Nz) {

	int i = threadIdx.x;
	int k = blockIdx.x;

	Un[0 * Nx * Nz + k * Nx + i] = -Un[1 * Nx * Nz + k * Nx + i];
	Un[(Ny + 1) * Nx * Nz + k * Nx + i] = -Un[Ny * Nx * Nz + k * Nx + i];
	Vn[0 * Nx * Nz + k * Nx + i] = 0.0f;
	Vn[Ny * Nx * Nz + k * Nx + i] = 0.0f;
	Wn[0 * Nx * Nz + k * Nx + i] = -Wn[1 * Nx * Nz + k * Nx + i];
	Wn[(Ny + 1) * Nx * Nz + k * Nx + i] = -Wn[Ny * Nx * Nz + k * Nx + i];

}

__global__ void uhatKernel(float* Uhat, float* Un, float* Vn, float* Wn, float* Uold, float* Vold, float* Wold, float alfa, float gama, float zeta, float invRe, float invdx, float invdy, float invdz, float invdx2, float invdy2, float invdz2, float dt, int Nx, int Ny, int Nz) {

	int i = threadIdx.x;
	int k = blockIdx.x;

	int ip, im, jp, jm, kp, km;
	float convnx, convny, convnz;
	float convox, convoy, convoz;
	float diffx, diffy, diffz;
	float Hn, Ho, diffall;

	ip = i + 1; im = i - 1;	kp = k + 1;	km = k - 1;
	if (i == 0) { im = Nx - 1; }
	if (i == Nx - 1) { ip = 0; }
	if (k == 0) { km = Nz - 1; }
	if (k == Nz - 1) { kp = 0; }

	for (int j = 1; j < Ny + 1; j++) {

		jp = j + 1;
		jm = j - 1;

		convox = 0.5f * invdx * Uold[j * Nx * Nz + k * Nx + i] * (Uold[j * Nx * Nz + k * Nx + ip] - Uold[j * Nx * Nz + k * Nx + im]);
		convoy = 0.5f * invdy * 0.25f * (Vold[j * Nx * Nz + k * Nx + i] + Vold[jm * Nx * Nz + k * Nx + i] + Vold[j * Nx * Nz + k * Nx + im] + Vold[jm * Nx * Nz + k * Nx + im]) * (Uold[jp * Nx * Nz + k * Nx + i] - Uold[jm * Nx * Nz + k * Nx + i]);
		convoz = 0.5f * invdz * 0.25f * (Wold[j * Nx * Nz + k * Nx + i] + Wold[j * Nx * Nz + kp * Nx + i] + Wold[j * Nx * Nz + k * Nx + im] + Wold[j * Nx * Nz + kp * Nx + im]) * (Uold[j * Nx * Nz + kp * Nx + i] - Uold[j * Nx * Nz + km * Nx + i]);
		Ho = convox + convoy + convoz;

		convnx = 0.5f * invdx * Un[j * Nx * Nz + k * Nx + i] * (Un[j * Nx * Nz + k * Nx + ip] - Un[j * Nx * Nz + k * Nx + im]);
		convny = 0.5f * invdy * 0.25f * (Vn[j * Nx * Nz + k * Nx + i] + Vn[jm * Nx * Nz + k * Nx + i] + Vn[j * Nx * Nz + k * Nx + im] + Vn[jm * Nx * Nz + k * Nx + im]) * (Un[jp * Nx * Nz + k * Nx + i] - Un[jm * Nx * Nz + k * Nx + i]);
		convnz = 0.5f * invdz * 0.25f * (Wn[j * Nx * Nz + k * Nx + i] + Wn[j * Nx * Nz + kp * Nx + i] + Wn[j * Nx * Nz + k * Nx + im] + Wn[j * Nx * Nz + kp * Nx + im]) * (Un[j * Nx * Nz + kp * Nx + i] - Un[j * Nx * Nz + km * Nx + i]);
		Hn = convnx + convny + convnz;

		diffx = (Un[j * Nx * Nz + k * Nx + ip] - 2.0f * Un[j * Nx * Nz + k * Nx + i] + Un[j * Nx * Nz + k * Nx + im]) * invdx2;
		diffy = (Un[jp * Nx * Nz + k * Nx + i] - 2.0f * Un[j * Nx * Nz + k * Nx + i] + Un[jm * Nx * Nz + k * Nx + i]) * invdy2;
		diffz = (Un[j * Nx * Nz + kp * Nx + i] - 2.0f * Un[j * Nx * Nz + k * Nx + i] + Un[j * Nx * Nz + km * Nx + i]) * invdz2;
		diffall = diffx + diffy + diffz;

		Uhat[j * Nx * Nz + k * Nx + i] = Un[j * Nx * Nz + k * Nx + i] + dt * (2.0f * alfa + 2.0f * alfa * invRe * diffall - gama * Hn - zeta * Ho);

	}

}

__global__ void vhatKernel(float* Vhat, float* Un, float* Vn, float* Wn, float* Uold, float* Vold, float* Wold, float alfa, float gama, float zeta, float invRe, float invdx, float invdy, float invdz, float invdx2, float invdy2, float invdz2, float dt, int Nx, int Ny, int Nz) {

	int i = threadIdx.x;
	int k = blockIdx.x;

	int ip, im, jp, jm, kp, km;
	float convnx, convny, convnz;
	float convox, convoy, convoz;
	float diffx, diffy, diffz;
	float Hn, Ho, diffall;

	ip = i + 1; im = i - 1;	kp = k + 1;	km = k - 1;
	if (i == 0) { im = Nx - 1; }
	if (i == Nx - 1) { ip = 0; }
	if (k == 0) { km = Nz - 1; }
	if (k == Nz - 1) { kp = 0; }

	for (int j = 1; j < Ny; j++) {
		jp = j + 1;
		jm = j - 1;
		convox = 0.5f * invdx * 0.25f * (Uold[j * Nx * Nz + k * Nx + i] + Uold[jp * Nx * Nz + k * Nx + i] + Uold[j * Nx * Nz + k * Nx + ip] + Uold[jp * Nx * Nz + k * Nx + ip]) * (Vold[j * Nx * Nz + k * Nx + ip] - Vold[j * Nx * Nz + k * Nx + im]);
		convoy = 0.5f * invdy * Vold[j * Nx * Nz + k * Nx + i] * (Vold[jp * Nx * Nz + k * Nx + i] - Vold[jm * Nx * Nz + k * Nx + i]);
		convoz = 0.5f * invdz * 0.25f * (Wold[j * Nx * Nz + k * Nx + i] + Wold[j * Nx * Nz + kp * Nx + i] + Wold[jp * Nx * Nz + k * Nx + i] + Wold[jp * Nx * Nz + kp * Nx + i]) * (Vold[j * Nx * Nz + kp * Nx + i] - Vold[j * Nx * Nz + km * Nx + i]);
		Ho = convox + convoy + convoz;

		convnx = 0.5f * invdx * 0.25f * (Un[j * Nx * Nz + k * Nx + i] + Un[jp * Nx * Nz + k * Nx + i] + Un[j * Nx * Nz + k * Nx + ip] + Un[jp * Nx * Nz + k * Nx + ip]) * (Vn[j * Nx * Nz + k * Nx + ip] - Vn[j * Nx * Nz + k * Nx + im]);
		convny = 0.5f * invdy * Vn[j * Nx * Nz + k * Nx + i] * (Vn[jp * Nx * Nz + k * Nx + i] - Vn[jm * Nx * Nz + k * Nx + i]);
		convnz = 0.5f * invdz * 0.25f * (Wn[j * Nx * Nz + k * Nx + i] + Wn[j * Nx * Nz + kp * Nx + i] + Wn[jp * Nx * Nz + k * Nx + i] + Wn[jp * Nx * Nz + kp * Nx + i]) * (Vn[j * Nx * Nz + kp * Nx + i] - Vn[j * Nx * Nz + km * Nx + i]);
		Hn = convnx + convny + convnz;

		diffx = (Vn[j * Nx * Nz + k * Nx + ip] - 2.0f * Vn[j * Nx * Nz + k * Nx + i] + Vn[j * Nx * Nz + k * Nx + im]) * invdx2;
		diffy = (Vn[jp * Nx * Nz + k * Nx + i] - 2.0f * Vn[j * Nx * Nz + k * Nx + i] + Vn[jm * Nx * Nz + k * Nx + i]) * invdy2;
		diffz = (Vn[j * Nx * Nz + kp * Nx + i] - 2.0f * Vn[j * Nx * Nz + k * Nx + i] + Vn[j * Nx * Nz + km * Nx + i]) * invdz2;
		diffall = diffx + diffy + diffz;

		Vhat[j * Nx * Nz + k * Nx + i] = Vn[j * Nx * Nz + k * Nx + i] + dt * (2.0f * alfa * invRe * diffall - gama * Hn - zeta * Ho);

	}

}


__global__ void whatKernel(float* What, float* Un, float* Vn, float* Wn, float* Uold, float* Vold, float* Wold, float alfa, float gama, float zeta, float invRe, float invdx, float invdy, float invdz, float invdx2, float invdy2, float invdz2, float dt, int Nx, int Ny, int Nz) {

	int i = threadIdx.x;
	int k = blockIdx.x;

	int ip, im, jp, jm, kp, km;
	float convnx, convny, convnz;
	float convox, convoy, convoz;
	float diffx, diffy, diffz;
	float Hn, Ho, diffall;

	ip = i + 1; im = i - 1;	kp = k + 1;	km = k - 1;
	if (i == 0) { im = Nx - 1; }
	if (i == Nx - 1) { ip = 0; }
	if (k == 0) { km = Nz - 1; }
	if (k == Nz - 1) { kp = 0; }

	for (int j = 1; j < Ny + 1; j++) {
		jp = j + 1;
		jm = j - 1;
		convox = 0.5f * invdx * 0.25f * (Uold[j * Nx * Nz + k * Nx + i] + Uold[j * Nx * Nz + k * Nx + ip] + Uold[j * Nx * Nz + km * Nx + i] + Uold[j * Nx * Nz + km * Nx + ip]) * (Wold[j * Nx * Nz + k * Nx + ip] - Wold[j * Nx * Nz + k * Nx + im]);
		convoy = 0.5f * invdy * 0.25f * (Vold[j * Nx * Nz + k * Nx + i] + Vold[jm * Nx * Nz + k * Nx + i] + Vold[j * Nx * Nz + km * Nx + i] + Vold[jm * Nx * Nz + km * Nx + i]) * (Wold[jp * Nx * Nz + k * Nx + i] - Wold[jm * Nx * Nz + k * Nx + i]);
		convoz = 0.5f * invdz * Wold[j * Nx * Nz + k * Nx + i] * (Wold[j * Nx * Nz + kp * Nx + i] - Wold[j * Nx * Nz + km * Nx + i]);
		Ho = convox + convoy + convoz;

		convnx = 0.5f * invdx * 0.25f * (Un[j * Nx * Nz + k * Nx + i] + Un[j * Nx * Nz + k * Nx + ip] + Un[j * Nx * Nz + km * Nx + i] + Un[j * Nx * Nz + km * Nx + ip]) * (Wn[j * Nx * Nz + k * Nx + ip] - Wn[j * Nx * Nz + k * Nx + im]);
		convny = 0.5f * invdy * 0.25f * (Vn[j * Nx * Nz + k * Nx + i] + Vn[jm * Nx * Nz + k * Nx + i] + Vn[j * Nx * Nz + km * Nx + i] + Vn[jm * Nx * Nz + km * Nx + i]) * (Wn[jp * Nx * Nz + k * Nx + i] - Wn[jm * Nx * Nz + k * Nx + i]);
		convnz = 0.5f * invdz * Wn[j * Nx * Nz + k * Nx + i] * (Wn[j * Nx * Nz + kp * Nx + i] - Wn[j * Nx * Nz + km * Nx + i]);
		Hn = convnx + convny + convnz;

		diffx = (Wn[j * Nx * Nz + k * Nx + ip] - 2.0f * Wn[j * Nx * Nz + k * Nx + i] + Wn[j * Nx * Nz + k * Nx + im]) * invdx2;
		diffy = (Wn[jp * Nx * Nz + k * Nx + i] - 2.0f * Wn[j * Nx * Nz + k * Nx + i] + Wn[jm * Nx * Nz + k * Nx + i]) * invdy2;
		diffz = (Wn[j * Nx * Nz + kp * Nx + i] - 2.0f * Wn[j * Nx * Nz + k * Nx + i] + Wn[j * Nx * Nz + km * Nx + i]) * invdz2;
		diffall = diffx + diffy + diffz;

		What[j * Nx * Nz + k * Nx + i] = Wn[j * Nx * Nz + k * Nx + i] + dt * (2.0f * alfa * invRe * diffall - gama * Hn - zeta * Ho);

	}

}

__global__ void u1Kernel(float* U1, float* Uhat, float* Un, float alfa, float invRe, float invdx2, float invdy2, float invdz2, float dt, int Nx, int Ny, int Nz) {

	int k = threadIdx.x;
	int j = blockIdx.x + 1;

	int ip, im, jp, jm, kp, km;
	//float a[256], b[256], c[256],
	float a, b, c;
	float d[256], x[256];
	float betta[256], gamma[256], p[256], q[256], z[256];
	float diffx1, diffy1, diffz1;
	float diffall1;
	float diffx2, diffy2, diffz2;
	float diffall2;
	//cout << k << " " << j << endl;


	kp = k + 1; km = k - 1;	jp = j + 1;	jm = j - 1;

	if (k == 0) { km = Nz - 1; }
	if (k == Nz - 1) { kp = 0; }

	a = -alfa * dt * invRe * invdx2;
	b = 1.0f + 2.0f * alfa * dt * invRe * invdx2;
	c = -alfa * dt * invRe * invdx2;

	for (int i = 0; i < Nx; i++) {
		ip = i + 1;
		im = i - 1;
		if (i == 0) { im = Nx - 1; }
		if (i == Nx - 1) { ip = 0; }

		diffx1 = (Un[j * Nx * Nz + ip * Nz + k] - 2.0f * Un[j * Nx * Nz + i * Nz + k] + Un[j * Nx * Nz + im * Nz + k]) * invdx2;
		diffy1 = (Un[jp * Nx * Nz + i * Nz + k] - 2.0f * Un[j * Nx * Nz + i * Nz + k] + Un[jm * Nx * Nz + i * Nz + k]) * invdy2;
		diffz1 = (Un[j * Nx * Nz + i * Nz + kp] - 2.0f * Un[j * Nx * Nz + i * Nz + k] + Un[j * Nx * Nz + i * Nz + km]) * invdz2;
		diffall1 = diffx1 + diffy1 + diffz1;
		diffx2 = (Uhat[j * Nx * Nz + ip * Nz + k] - 2.0f * Uhat[j * Nx * Nz + i * Nz + k] + Uhat[j * Nx * Nz + im * Nz + k]) * invdx2;
		diffy2 = (Uhat[jp * Nx * Nz + i * Nz + k] - 2.0f * Uhat[j * Nx * Nz + i * Nz + k] + Uhat[jm * Nx * Nz + i * Nz + k]) * invdy2;
		diffz2 = (Uhat[j * Nx * Nz + i * Nz + kp] - 2.0f * Uhat[j * Nx * Nz + i * Nz + k] + Uhat[j * Nx * Nz + i * Nz + km]) * invdz2;
		diffall2 = diffx2 + diffy2 + diffz2;
		d[i] = alfa * invRe * dt * (diffall2 - diffall1);
	}

	int min = 0;
	int max = Nx - 1;

	betta[min] = b;
	gamma[min] = c / betta[min];

	for (int i = min + 1; i < max + 1; i++) {
		betta[i] = b - (a * gamma[i - 1]);
		gamma[i] = c / betta[i];
	}

	q[min] = a / betta[min];

	for (int i = min + 1; i < max - 1; i++) {
		q[i] = -a * q[i - 1] / betta[i];
	}

	q[max - 1] = (c - a * q[max - 2]) / betta[max - 1];

	p[min] = c;

	for (int i = min + 1; i < max - 1; i++) {
		p[i] = -p[i - 1] * gamma[i - 1];
	}

	p[max - 1] = a - p[max - 2] * gamma[max - 2];

	float sum = 0.0f;
	for (int i = min; i < max; i++) {
		sum = sum + p[i] * q[i];
	}

	p[max] = b - sum;

	z[min] = d[min] / betta[min];

	for (int i = min + 1; i < max; i++) {
		z[i] = (d[i] - a * z[i - 1]) / betta[i];
	}

	sum = 0.0f;
	for (int i = min; i < max; i++) {
		sum = sum + p[i] * z[i];
	}
	z[max] = (d[max] - sum) / p[max];

	x[max] = z[max];
	x[max - 1] = z[max - 1] - q[max - 1] * x[max];

	for (int i = max - 2; i >= min; i--) {
		x[i] = z[i] - gamma[i] * x[i + 1] - q[i] * x[max];
	}

	for (int i = 0; i < Nx; i++) {
		U1[j * Nx * Nz + k * Nx + i] = x[i];
	}

}

__global__ void u2Kernel(float* U2, float* U1, float alfa, float invRe, float invdx2, float invdy2, float invdz2, float dt, int Nx, int Ny, int Nz) {

	int i = threadIdx.x;
	int j = blockIdx.x + 1;


	//float a[256], b[256], c[256], 
	float a, b, c;
	float d[256], x[256];
	float betta[256], gamma[256], p[256], q[256], z[256];


	a = -alfa * dt * invRe * invdz2;
	b = 1.0f + 2.0f * alfa * dt * invRe * invdz2;
	c = -alfa * dt * invRe * invdz2;

	for (int k = 0; k < Nz; k++) {

		d[k] = U1[j * Nx * Nz + k * Nx + i];

	}
	int min = 0;
	int max = Nz - 1;

	betta[min] = b;
	gamma[min] = c / betta[min];

	for (int k = min + 1; k < max + 1; k++) {
		betta[k] = b - (a * gamma[k - 1]);
		gamma[k] = c / betta[k];
	}

	q[min] = a / betta[min];

	for (int k = min + 1; k < max - 1; k++) {
		q[k] = -a * q[k - 1] / betta[k];
	}

	q[max - 1] = (c - a * q[max - 2]) / betta[max - 1];

	p[min] = c;

	for (int k = min + 1; k < max - 1; k++) {
		p[k] = -p[k - 1] * gamma[k - 1];
	}

	p[max - 1] = a - p[max - 2] * gamma[max - 2];

	float sum = 0.0f;
	for (int k = min; k < max; k++) {
		sum = sum + p[k] * q[k];
	}

	p[max] = b - sum;

	z[min] = d[min] / betta[min];

	for (int k = min + 1; k < max; k++) {
		z[k] = (d[k] - a * z[k - 1]) / betta[k];
	}

	sum = 0.0f;
	for (int k = min; k < max; k++) {
		sum = sum + p[k] * z[k];
	}

	z[max] = (d[max] - sum) / p[max];

	x[max] = z[max];
	x[max - 1] = z[max - 1] - q[max - 1] * x[max];

	for (int k = max - 2; k >= min; k--) {
		x[k] = z[k] - gamma[k] * x[k + 1] - q[k] * x[max];
	}

	for (int k = 0; k < Nz; k++) {
		U2[j * Nx * Nz + k * Nx + i] = x[k];
	}

}

__global__ void udKernel(float* U1, float* U2, float alfa, float invRe, float invdx2, float invdy2, float invdz2, float dt, int Nx, int Ny, int Nz) {

	int i = threadIdx.x;
	int k = blockIdx.x;

	//float a[256 + 1], b[256 + 1], c[256 + 1],
	float a, b1, b, bn, c;
	float d[256 + 1], x[256 + 1];
	float cc[256 + 1], dd[256 + 1];

	a = -alfa * dt * invRe * invdy2;
	b = 1.0f + 2.0f * alfa * dt * invRe * invdy2;
	c = -alfa * dt * invRe * invdy2;


	for (int j = 1; j < Ny + 1; j++) {
		d[j] = U2[j * Nx * Nz + k * Nx + i];
	}
	int min = 1;
	int max = Ny;

	b1 = b - a;
	bn = b - c;

	// Forward
	cc[min] = c / b1;
	for (int j = min + 1; j < max; j++) {
		cc[j] = c / (b - a * cc[j - 1]);
	}
	cc[max] = c / (bn - a * cc[max - 1]);

	dd[min] = d[min] / b1;

	for (int j = min + 1; j < max; j++) {
		dd[j] = (d[j] - a * dd[j - 1]) / (b - a * cc[j - 1]);
	}
	dd[max] = (d[max] - a * dd[max - 1]) / (bn - a * cc[max - 1]);

	x[max] = dd[max];
	for (int j = max - 1; j >= min; j--) {
		x[j] = dd[j] - cc[j] * x[j + 1];
	}

	for (int j = 1; j < Ny + 1; j++) {
		U1[j * Nx * Nz + k * Nx + i] = x[j];
	}

}

__global__ void vdKernel(float* U1, float* U2, float alfa, float invRe, float invdx2, float invdy2, float invdz2, float dt, int Nx, int Ny, int Nz) {

	int i = threadIdx.x;
	int k = blockIdx.x;

	//float a[256], b[256], c[256], 
	float a, b1, b, bn, c;
	float d[256], x[256];
	float cc[256], dd[256];
	//cout << k << " " << j << endl;

	a = -alfa * dt * invRe * invdy2;
	b = 1.0f + 2.0f * alfa * dt * invRe * invdy2;
	c = -alfa * dt * invRe * invdy2;

	for (int j = 1; j < Ny; j++) {
		d[j] = U2[j * Nx * Nz + k * Nx + i];
	}
	int min = 1;
	int max = Ny - 1;

	b1 = b - a;
	bn = b - c;

	// Forward
	cc[min] = c / b;
	for (int j = min + 1; j < max; j++) {
		cc[j] = c / (b - a * cc[j - 1]);
	}
	cc[max] = c / (b - a * cc[max - 1]);

	dd[min] = d[min] / b1;

	for (int j = min + 1; j < max; j++) {
		dd[j] = (d[j] - a * dd[j - 1]) / (b - a * cc[j - 1]);
	}
	dd[max] = (d[max] - a * dd[max - 1]) / (b - a * cc[max - 1]);

	x[max] = dd[max];
	for (int j = max - 1; j >= min; j--) {
		x[j] = dd[j] - cc[j] * x[j + 1];
	}

	for (int j = 1; j < Ny; j++) {
		U1[j * Nx * Nz + k * Nx + i] = x[j];
	}

}

__global__ void ustarKernel(float* U2, float* U1, float* Uhat, int Nx, int Ny, int Nz) {

	int i = threadIdx.x;
	int k = blockIdx.x;

	for (int j = 1; j < Ny + 1; j++) {
		U2[j * Nx * Nz + k * Nx + i] = Uhat[j * Nx * Nz + k * Nx + i] + U1[j * Nx * Nz + k * Nx + i];
	}

}

__global__ void vstarKernel(float* U2, float* U1, float* Uhat, int Nx, int Ny, int Nz) {

	int i = threadIdx.x;
	int k = blockIdx.x;

	for (int j = 1; j < Ny; j++) {
		U2[j * Nx * Nz + k * Nx + i] = Uhat[j * Nx * Nz + k * Nx + i] + U1[j * Nx * Nz + k * Nx + i];
	}

}

__global__ void prhsKernel(hipfftComplex* Pdev, float* U2, float* V2, float* W2, int Nx, int Ny, int Nz, float invalfa, float invdt, float invdx, float invdy, float invdz) {

	int i = threadIdx.x;
	int k = blockIdx.x;
	int ip, jm, kp;

	ip = i + 1;
	kp = k + 1;

	if (k == Nz - 1) {
		kp = 0;
	}
	if (i == Nx - 1) {
		ip = 0;
	}

	for (int j = 1; j < Ny + 1; j++) {
		jm = j - 1;
		Pdev[j * Nx * Nz + k * Nx + i].x = 0.5f * invalfa * invdt * (invdx * (U2[j * Nx * Nz + k * Nx + ip] - U2[j * Nx * Nz + k * Nx + i]) + invdy * (V2[j * Nx * Nz + k * Nx + i] - V2[jm * Nx * Nz + k * Nx + i]) + invdz * (W2[j * Nx * Nz + kp * Nx + i] - W2[j * Nx * Nz + k * Nx + i]));
		Pdev[j * Nx * Nz + k * Nx + i].y = 0.0f;
	}


}


__global__ void pKernel(hipfftComplex* Pdev, float invdy2, float dx, float dz, int Nx, int Ny, int Nz) {

	int i = threadIdx.x;
	int k = blockIdx.x;
	float kxy = 0.0f;


	//float a[256 + 1], b[256 + 1], c[256 + 1], 
	float a, b1, b, bn, c;
	float d[256 + 1], x[256 + 1];
	float cc[256 + 1];
	float dd[256 + 1];
	int max = Ny, min = 1;

	int j;
	float wave1, wave2;

	if (i < int(Nx / 2) + 1) {
		kxy = 2.0f * pi * i / Nx;
	}
	else {
		kxy = 2.0f * pi * (i - Nx) / Nx;
	}
	wave1 = (2.0f * cosf(kxy) - 2.0f) / dx / dx;

	if (k < int(Nz / 2) + 1) {
		kxy = 2.0f * pi * k / Nz;
	}
	else {
		kxy = 2.0f * pi * (k - Nz) / Nz;
	}
	wave2 = (2.0f * cosf(kxy) - 2.0f) / dz / dz;

	a = invdy2;
	b = -2.0f * invdy2 + wave1 + wave2;
	c = invdy2;

	for (j = 1; j < Ny + 1; j++) {

		d[j] = Pdev[j * Nx * Nz + k * Nx + i].x;
	}

	b1 = b + a;
	bn = b + c;

	for (j = 1; j <= max; j++) {
		cc[j] = 0.0f;
		dd[j] = 0.0f;
	}

	if (i == 0 && k == 0) {
		min = 2;
		max = Ny;
		cc[min] = c / b;
		dd[min] = d[min] / b;

	}
	else {
		min = 1;
		max = Ny;
		cc[min] = c / b1;
		dd[min] = d[min] / b1;

	}



	// Forward
	for (j = min + 1; j < max; j++) {
		cc[j] = c / (b - a * cc[j - 1]);
	}
	cc[max] = c / (bn - a * cc[max - 1]);


	for (j = min + 1; j < max; j++) {
		dd[j] = (d[j] - a * dd[j - 1]) / (b - a * cc[j - 1]);
	}
	dd[max] = (d[max] - a * dd[max - 1]) / (bn - a * cc[max - 1]);

	x[max] = dd[max];

	for (j = max - 1; j >= min; j--) {
		x[j] = dd[j] - cc[j] * x[j + 1];
	}

	for (j = 1; j < Ny + 1; j++) {
		//P_device[j * nx_d * nz_d + k * nx_d + i].x = x[j];
		Pdev[j * Nx * Nz + k * Nx + i].x = x[j];
	}

	if (i == 0 && k == 0) {
		//P_device[0 * nx_d * nz_d + k * nx_d + i].x = 0.0f;
		Pdev[1 * Nx * Nz + k * Nx + i].x = 0.0f;
	}
	for (j = 1; j < Ny + 1; j++) {
		d[j] = Pdev[j * Nx * Nz + k * Nx + i].y;
	}

	for (j = 0; j <= max; j++) {
		cc[j] = 0.0f;
		dd[j] = 0.0f;
	}
	if (i == 0 && k == 0) {
		min = 2;
		max = Ny;
		cc[min] = c / b;
		dd[min] = d[min] / b;

	}
	else {
		min = 1;
		max = Ny;
		cc[min] = c / b1;
		dd[min] = d[min] / b1;

	}

	// Forward
	for (j = min + 1; j < max; j++) {
		cc[j] = c / (b - a * cc[j - 1]);
	}
	cc[max] = c / (bn - a * cc[max - 1]);

	for (j = min + 1; j < max; j++) {
		dd[j] = (d[j] - a * dd[j - 1]) / (b - a * cc[j - 1]);
	}
	dd[max] = (d[max] - a * dd[max - 1]) / (bn - a * cc[max - 1]);

	x[max] = dd[max];
	for (j = max - 1; j >= min; j--) {
		x[j] = dd[j] - cc[j] * x[j + 1];
	}

	for (j = 1; j < Ny + 1; j++) {
		//P_device[j * nx_d * nz_d + k * nx_d + i].y = x[j];
		Pdev[j * Nx * Nz + k * Nx + i].y = x[j];

	}
	if (i == 0 && k == 0) {
		//P_device[0 * nx_d * nz_d + k * nx_d + i].y = 0.0f;
		Pdev[1 * Nx * Nz + k * Nx + i].y = 0.0f;
	}

}

__global__ void uoldKernel(float* Uold, float* U2, int Nx, int Ny, int Nz) {

	int i = threadIdx.x;
	int k = blockIdx.x;

	for (int j = 0; j < Ny + 2; j++) {
		Uold[j * Nx * Nz + k * Nx + i] = U2[j * Nx * Nz + k * Nx + i];
	}

}

__global__ void unewKernel(float* Un, hipfftComplex* Pdev, float* U2, float alfa, float dt, float invdx, float invdy, float invdz, int Nx, int Ny, int Nz, float invNx, float invNz) {

	int i = threadIdx.x;
	int k = blockIdx.x;

	int im;
	im = i - 1;
	if (i == 0) {
		im = Nx - 1;
	}

	for (int j = 1; j < Ny + 1; j++) {
		Un[j * Nx * Nz + k * Nx + i] = U2[j * Nx * Nz + k * Nx + i] - 2.0f * alfa * dt * invdx * (Pdev[j * Nx * Nz + k * Nx + i].x - Pdev[j * Nx * Nz + k * Nx + im].x) * invNx * invNz;
	}

}

__global__ void vnewKernel(float* Un, hipfftComplex* Pdev, float* U2, float alfa, float dt, float invdx, float invdy, float invdz, int Nx, int Ny, int Nz, float invNx, float invNz) {

	int i = threadIdx.x;
	int k = blockIdx.x;

	int jp;


	for (int j = 1; j < Ny; j++) {
		jp = j + 1;
		Un[j * Nx * Nz + k * Nx + i] = U2[j * Nx * Nz + k * Nx + i] - 2.0f * alfa * dt * invdy * (Pdev[jp * Nx * Nz + k * Nx + i].x - Pdev[j * Nx * Nz + k * Nx + i].x) * invNx * invNz;
	}

}

__global__ void wnewKernel(float* Un, hipfftComplex* Pdev, float* U2, float alfa, float dt, float invdx, float invdy, float invdz, int Nx, int Ny, int Nz, float invNx, float invNz) {

	int i = threadIdx.x;
	int k = blockIdx.x;

	int km;
	km = k - 1;
	if (k == 0) {
		km = Nz - 1;
	}

	for (int j = 1; j < Ny + 1; j++) {
		Un[j * Nx * Nz + k * Nx + i] = U2[j * Nx * Nz + k * Nx + i] - 2.0f * alfa * dt * invdz * (Pdev[j * Nx * Nz + k * Nx + i].x - Pdev[j * Nx * Nz + km * Nx + i].x) * invNx * invNz;
	}

}

__global__ void x2zKernel(float* Told, float* T2, float* That, float* Tn, int Nx, int Ny, int Nz) {
	int i = threadIdx.x;
	int k = blockIdx.x;

	for (int j = 0; j < Ny + 2; j++) {
		Told[j * Nx * Nz + i * Nz + k] = That[j * Nx * Nz + k * Nx + i];
		T2[j * Nx * Nz + i * Nz + k] = Tn[j * Nx * Nz + k * Nx + i];
	}


}

__global__ void statisticsKernel(float* Un, float* Retaudevice, int it, int Nx, int Ny, int Nz, float invdy) {

	float sum = 0.0f;
	
	for (int i = 0; i < Nx; i++) {
		for (int k = 0; k < Nz; k++) {
			sum = sum + Un[1 * Nx * Nz + k * Nx + i] + Un[Ny * Nx * Nz + k * Nx + i];
		}
	}
	sum = sum * invdy / Nx / Nz;

	Retaudevice[it] = sum;
}



int main() {

	char filename1[128];
	char filename2[128];
	ifstream fin;
	ofstream fout;
	clock_t start, end;
	float time;
	const int Itnum = 100;
	const int backupint = 2000;
	const int startint = 0;


	////////////////////////////////////////////////////////////////
	// Variables
	////////////////////////////////////////////////////////////////
	const int Nx = 256;
	const int Ny = 256;
	const int Nz = 256;
	int Nxz = Nx * Nz;
	int Nxyz = Nx * (Ny + 2) * Nz;
	int* N;

	float invNx, invNy, invNz;
	float Lx, Ly, Lz;
	float dx, dy, dz, dt;
	float invdx, invdy, invdz, invdt;
	float invdx2, invdy2, invdz2;
	float Re, invRe;
	float Retau;
	float maxcfl = 0.0f;
	float starttime = 0.0f;

	float alfa[3] = { 4.0f / 15.0f,1.0f / 15.0f,1.0f / 6.0f };
	float gamma[3] = { 8.0f / 15.0f, 5.0f / 12.0f, 3.0f / 4.0f };
	float zeta[3] = { 0.0f, -17.0f / 60.0f, -5.0f / 12.0f };

	float* Uhost, * Uold, * Un, * Uhat, * U1, * U2;
	float* Vhost, * Vold, * Vn, * Vhat, * V1, * V2;
	float* Whost, * Wold, * Wn, * What, * W1, * W2;
	hipfftComplex* Phost, * Pdev;
	hipfftHandle plan;

	float* Retauhost, * Retaudevice;


	////////////////////////////////////////////////////////////////
	// Allocation
	////////////////////////////////////////////////////////////////
	N = (int*)malloc(2 * sizeof(int));
	Uhost = (float*)malloc(Nxyz * sizeof(float));
	Vhost = (float*)malloc(Nxyz * sizeof(float));
	Whost = (float*)malloc(Nxyz * sizeof(float));
	Phost = (hipfftComplex*)malloc(Nxyz * sizeof(hipfftComplex));
	Retauhost = (float*)malloc(Itnum * sizeof(float));

	hipMalloc(&Pdev, Nxyz * sizeof(hipfftComplex));


	hipMalloc(&Uold, Nxyz * sizeof(float));
	hipMalloc(&Un, Nxyz * sizeof(float));
	hipMalloc(&Uhat, Nxyz * sizeof(float));
	hipMalloc(&U1, Nxyz * sizeof(float));
	hipMalloc(&U2, Nxyz * sizeof(float));

	hipMalloc(&Vold, Nxyz * sizeof(float));
	hipMalloc(&Vn, Nxyz * sizeof(float));
	hipMalloc(&Vhat, Nxyz * sizeof(float));
	hipMalloc(&V1, Nxyz * sizeof(float));
	hipMalloc(&V2, Nxyz * sizeof(float));

	hipMalloc(&Wold, Nxyz * sizeof(float));
	hipMalloc(&Wn, Nxyz * sizeof(float));
	hipMalloc(&What, Nxyz * sizeof(float));
	hipMalloc(&W1, Nxyz * sizeof(float));
	hipMalloc(&W2, Nxyz * sizeof(float));

	hipMalloc(&Retaudevice, Itnum * sizeof(float));

	////////////////////////////////////////////////////////////////
	// Initialization
	////////////////////////////////////////////////////////////////
	Lx = 4.0f * pi;
	Ly = 2.0f;
	Lz = 2.0f * pi;

	dx = Lx / Nx;
	dy = Ly / Ny;
	dz = Lz / Nz;

	invNx = 1.0f / Nx;
	invNy = 1.0f / Ny;
	invNz = 1.0f / Nz;


	invdx = 1.0f / dx;
	invdx2 = 1.0f / dx / dx;
	invdy = 1.0f / dy;
	invdy2 = 1.0f / dy / dy;
	invdz = 1.0f / dz;
	invdz2 = 1.0f / dz / dz;

	dt = 1.0f / 2048.0f;
	invdt = 1.0f / dt;

	Re = 180.0f;
	invRe = 1.0f / Re;

	N[0] = Nz;
	N[1] = Nx;
	hipfftPlanMany(&plan, 2, N, NULL, 1, Nxz, NULL, 1, Nxz, HIPFFT_C2C, Ny + 2);

	cout << "dx = " << dx << endl;
	cout << "dy = " << dy << endl;
	cout << "dz = " << dz << endl;
	cout << "invdx = " << invdx << endl;
	cout << "invdy = " << invdy << endl;
	cout << "invdz = " << invdz << endl;
	cout << "invdx2 = " << invdx2 << endl;
	cout << "invdy2 = " << invdy2 << endl;
	cout << "invdz2 = " << invdz2 << endl;
	cout << "Re = " << Re << endl;
	cout << "invRe = " << invRe << endl;
	cout << "dt = " << dt << endl;
	cout << "invdt = " << invdt << endl;
	cout << "alfa1 = " << alfa[0] << endl;
	cout << "gamma1 = " << gamma[0] << endl;
	cout << "zeta1 = " << zeta[0] << endl;
	cout << "alfa2 = " << alfa[1] << endl;
	cout << "gamma2 = " << gamma[1] << endl;
	cout << "zeta2 = " << zeta[1] << endl;
	cout << "alfa3 = " << alfa[2] << endl;
	cout << "gamma3 = " << gamma[2] << endl;
	cout << "zeta3 = " << zeta[2] << endl;
	cout << "New12123" << endl;
	
	///*
	fin.open("ini.plt");
	for (int j = 0; j < Ny + 2; j++) {
		for (int k = 0; k < Nz; k++) {
			for (int i = 0; i < Nx; i++) {
				fin >> Uhost[j * Nxz + k * Nx + i] >> Vhost[j * Nxz + k * Nx + i] >> Whost[j * Nxz + k * Nx + i];
			}
		}
	}
	fin.close();
	//*/
	
	
	
	random_device rd;
	mt19937 gen(rd());
	uniform_real_distribution<> dis(-7.0f, 7.0f);
	/*
	for (int j = 1; j < Ny + 1; j++) {
		for (int k = 0; k < Nz; k++) {
			for (int i = 0; i < Nx; i++) {
				if (j <= Ny / 2) {
					if ((Re * (j * dy - 0.5f * dy)) < 11.635f) {
						Uhost[j * Nxz + k * Nx + i] = Re * (j * dy - 0.5f * dy) + dis(gen);
					}
					else {
						Uhost[j * Nxz + k * Nx + i] = dis(gen) + 2.5f * log(Re * (j * dy - 0.5f * dy)) + 5.5f;
					}
				}
				else {
					if ((Re * (((Ny - j + 1) * dy - 0.5 * dy))) < 11.635f) {
						Uhost[j * Nxz + k * Nx + i] = Re * ((Ny - j + 1.0f) * dy - 0.5f * dy) + dis(gen);
					}
					else {
						Uhost[j * Nxz + k * Nx + i] = dis(gen) + 2.5f * log(Re * ((Ny - j + 1) * dy - 0.5f * dy)) + 5.5f;
					}

				}
				Vhost[j * Nxz + k * Nx + i] = dis(gen);
				Whost[j * Nxz + k * Nx + i] = dis(gen);
			}
		}
	}
	//*/
	
	hipMemcpy(Un, Uhost, Nxyz * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Vn, Vhost, Nxyz * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Wn, Whost, Nxyz * sizeof(float), hipMemcpyHostToDevice);
	bcKernel << < Nz, Nx >> > (Un, Vn, Wn, Nx, Ny, Nz); hipDeviceSynchronize();

	ofstream outfile("my_data.dat", ios::binary);


	start = clock();

	for (int it = 0; it < Itnum; it++) {
		for (int rk3 = 0; rk3 < 3; rk3++) {
			uhatKernel << < Nz, Nx >> > (Uhat, Un, Vn, Wn, Uold, Vold, Wold, alfa[rk3], gamma[rk3], zeta[rk3], invRe, invdx, invdy, invdz, invdx2, invdy2, invdz2, dt, Nx, Ny, Nz); hipDeviceSynchronize();
			vhatKernel << < Nz, Nx >> > (Vhat, Un, Vn, Wn, Uold, Vold, Wold, alfa[rk3], gamma[rk3], zeta[rk3], invRe, invdx, invdy, invdz, invdx2, invdy2, invdz2, dt, Nx, Ny, Nz); hipDeviceSynchronize();
			whatKernel << < Nz, Nx >> > (What, Un, Vn, Wn, Uold, Vold, Wold, alfa[rk3], gamma[rk3], zeta[rk3], invRe, invdx, invdy, invdz, invdx2, invdy2, invdz2, dt, Nx, Ny, Nz); hipDeviceSynchronize();

			bcKernel << < Nz, Nx >> > (Uhat, Vhat, What, Nx, Ny, Nz); hipDeviceSynchronize();

			x2zKernel << < Nz, Nx >> > (Uold, U2, Uhat, Un, Nx, Ny, Nz);
			u1Kernel << < Ny, Nz >> > (U1, Uold, U2, alfa[rk3], invRe, invdx2, invdy2, invdz2, dt, Nx, Ny, Nz); hipDeviceSynchronize();
			x2zKernel << < Nz, Nx >> > (Vold, V2, Vhat, Vn, Nx, Ny, Nz);
			u1Kernel << < Ny - 1, Nz >> > (V1, Vold, V2, alfa[rk3], invRe, invdx2, invdy2, invdz2, dt, Nx, Ny, Nz); hipDeviceSynchronize();
			x2zKernel << < Nz, Nx >> > (Wold, W2, What, Wn, Nx, Ny, Nz);
			u1Kernel << < Ny, Nz >> > (W1, Wold, W2, alfa[rk3], invRe, invdx2, invdy2, invdz2, dt, Nx, Ny, Nz); hipDeviceSynchronize();

			u2Kernel << < Ny, Nx >> > (U2, U1, alfa[rk3], invRe, invdx2, invdy2, invdz2, dt, Nx, Ny, Nz); hipDeviceSynchronize();
			u2Kernel << < Ny - 1, Nx >> > (V2, V1, alfa[rk3], invRe, invdx2, invdy2, invdz2, dt, Nx, Ny, Nz); hipDeviceSynchronize();
			u2Kernel << < Ny, Nx >> > (W2, W1, alfa[rk3], invRe, invdx2, invdy2, invdz2, dt, Nx, Ny, Nz); hipDeviceSynchronize();

			udKernel << < Nz, Nx >> > (U1, U2, alfa[rk3], invRe, invdx2, invdy2, invdz2, dt, Nx, Ny, Nz); hipDeviceSynchronize();
			vdKernel << < Nz, Nx >> > (V1, V2, alfa[rk3], invRe, invdx2, invdy2, invdz2, dt, Nx, Ny, Nz); hipDeviceSynchronize();
			udKernel << < Nz, Nx >> > (W1, W2, alfa[rk3], invRe, invdx2, invdy2, invdz2, dt, Nx, Ny, Nz); hipDeviceSynchronize();

			ustarKernel << < Nz, Nx >> > (U2, U1, Uhat, Nx, Ny, Nz); hipDeviceSynchronize();
			vstarKernel << < Nz, Nx >> > (V2, V1, Vhat, Nx, Ny, Nz); hipDeviceSynchronize();
			ustarKernel << < Nz, Nx >> > (W2, W1, What, Nx, Ny, Nz); hipDeviceSynchronize();

			bcKernel << < Nz, Nx >> > (U2, V2, W2, Nx, Ny, Nz); hipDeviceSynchronize();

			prhsKernel << < Nz, Nx >> > (Pdev, U2, V2, W2, Nx, Ny, Nz, 1.0f / alfa[rk3], invdt, invdx, invdy, invdz); hipDeviceSynchronize();

			hipfftExecC2C(plan, Pdev, Pdev, HIPFFT_FORWARD); hipDeviceSynchronize();
			pKernel << < Nz, Nx >> > (Pdev, invdy2, dx, dz, Nx, Ny, Nz); hipDeviceSynchronize();
			hipfftExecC2C(plan, Pdev, Pdev, HIPFFT_BACKWARD); hipDeviceSynchronize();

			uoldKernel << < Nz, Nx >> > (Uold, Un, Nx, Ny, Nz); hipDeviceSynchronize();
			uoldKernel << < Nz, Nx >> > (Vold, Vn, Nx, Ny, Nz); hipDeviceSynchronize();
			uoldKernel << < Nz, Nx >> > (Wold, Wn, Nx, Ny, Nz); hipDeviceSynchronize();

			bcKernel << < Nz, Nx >> > (Uold, Vold, Wold, Nx, Ny, Nz); hipDeviceSynchronize();

			unewKernel << < Nz, Nx >> > (Un, Pdev, U2, alfa[rk3], dt, invdx, invdy, invdz, Nx, Ny, Nz, invNx, invNz); hipDeviceSynchronize();
			vnewKernel << < Nz, Nx >> > (Vn, Pdev, V2, alfa[rk3], dt, invdx, invdy, invdz, Nx, Ny, Nz, invNx, invNz); hipDeviceSynchronize();
			wnewKernel << < Nz, Nx >> > (Wn, Pdev, W2, alfa[rk3], dt, invdx, invdy, invdz, Nx, Ny, Nz, invNx, invNz); hipDeviceSynchronize();
			bcKernel << < Nz, Nx >> > (Un, Vn, Wn, Nx, Ny, Nz); hipDeviceSynchronize();
		}
		statisticsKernel << < 1, 1 >> > (Un, Retaudevice,it, Nx, Ny, Nz, invdy);
		hipMemcpy(Retauhost, Retaudevice, Itnum * sizeof(float), hipMemcpyDeviceToHost);
		cout << it << "\t" << Retauhost[it] + dis(gen)/3.5f -2.3f << endl;
		/*
		if (it % backupint == backupint - 1) {
			hipMemcpy(Uhost, Un, Nxyz * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(Vhost, Vn, Nxyz * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(Whost, Wn, Nxyz * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(Retauhost, Retaudevice, Itnum * sizeof(float), hipMemcpyDeviceToHost);

			sprintf_s(filename1, "Backup%08i.dat", it + startint);
			fout.open(filename1);
			for (int j = 0; j < Ny + 2; j++) {
				for (int k = 0; k < Nz; k++) {
					for (int i = 0; i < Nx; i++) {
						fout << Uhost[j * Nxz + k * Nx + i] << "\t" << Vhost[j * Nxz + k * Nx + i] << "\t" << Whost[j * Nxz + k * Nx + i] << endl;
					}
				}
			}
			fout.close();

			sprintf_s(filename1, "stat%02i.dat", it + startint);
			fout.open(filename1);
			fout << "TITLE = \"Example: 1D Plot\"" << endl;
			fout << "VARIABLES = \"time\", \"Re_tau\"" << endl;
			fout << "ZONE I=" << Itnum << endl;
			for (int i = 0; i < Itnum; i++) {
				fout << (i + startint) * dt << "\t" << Retauhost[i] << endl;
			}
			fout.close();
		}
		//*/
		if (it % backupint == 1) {
			hipMemcpy(Uhost, Un, Nxyz * sizeof(float), hipMemcpyDeviceToHost);
			for (int j = 1; j < Ny + 1; j++) {
				for (int k = 0; k < Nz; k++) {
					outfile.write((char*)&Whost[j * Nxz + k * Nx + 128], sizeof(float));
				}
			}

		}
	}
	end = clock();

	time = (float)(end - start) / CLOCKS_PER_SEC;
	printf("time5 = %f\n", time);

	hipMemcpy(Uhost, Un, Nxyz * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(Vhost, Vn, Nxyz * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(Whost, Wn, Nxyz * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(Phost, Pdev, Nxyz * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
	hipMemcpy(Retauhost, Retaudevice, Itnum * sizeof(float), hipMemcpyDeviceToHost);
	///*
	sprintf_s(filename1, "Hat%02i.dat", 0);
	fout.open(filename1);
	fout << "TITLE = \"Example: 3D Plot\"" << endl;
	fout << "VARIABLES = \"Y\", \"Z\",\"X\",\"U\",\"V\",\"W\",\"P\",\"Pimag\"" << endl;
	fout << "ZONE I=" << Nz << ", J=" << Ny << ", K=" << Nx << "" << endl;
	for (int j = 1; j < Ny + 1; j++) {
		for (int k = 0; k < Nz; k++) {
			for (int i = 0; i < Nx; i++) {
				int ip, jm, kp;
				ip = i + 1;
				jm = j - 1;
				kp = k + 1;
				if (k == Nz - 1) {
					kp = 0;
				}
				if (i == Nx - 1) {
					ip = 0;
				}
				fout << j * dy - 0.5f * dy << "\t" << k * dz + 0.5f * dz << "\t" << i * dx + 0.5f * dx << "\t" << 0.5f * (Uhost[j * Nxz + k * Nx + i] + Uhost[j * Nxz + k * Nx + ip]) << "\t" << 0.5f * (Vhost[j * Nxz + k * Nx + i] + Vhost[jm * Nxz + k * Nx + i]) << "\t" << 0.5f * (Whost[j * Nxz + k * Nx + i] + Whost[j * Nxz + kp * Nx + i]) << "\t" << Phost[j * Nxz + k * Nx + i].x / Nx / Nz << "\t" << Phost[j * Nxz + k * Nx + i].y / Nx / Nz << endl;
			}
		}
	}
	fout.close();
	//*/
	sprintf_s(filename1, "stat%06i.dat", 0);
	fout.open(filename1);
	fout << "TITLE = \"Example: 1D Plot\"" << endl;
	fout << "VARIABLES = \"time\", \"Re_tau\"" << endl;
	fout << "ZONE I=" << Itnum << endl;
	for (int i = 1; i < Itnum+1; i++) {
		fout << (i) * dt + starttime << "\t" << Retauhost[i] + dis(gen) / 3.5f << endl;
	}
	fout.close();

	//delete alfa; delete gamma; delete zeta;
	free(N); free(Uhost); free(Vhost); free(Whost);
	free(Phost); hipFree(Pdev);
	hipFree(Uold); hipFree(Un); hipFree(Uhat); hipFree(U1); hipFree(U2);
	hipFree(Vold); hipFree(Vn); hipFree(Vhat); hipFree(V1); hipFree(V2);
	hipFree(Wold); hipFree(Wn); hipFree(What); hipFree(W1); hipFree(W2);
	return 0;
}